#include "hip/hip_runtime.h"
#include "mops/hpe.hpp"

#include "internal/checks.hpp"
#include "internal/cuda_utils.cuh"

using namespace mops;
using namespace mops::cuda;

#define WARP_SIZE 32
#define NWARPS_PER_BLOCK 4

#define FULL_MASK 0xffffffff

using namespace mops;
using namespace mops::cuda;

template <typename scalar_t, int32_t polynomial_order>
__global__ void homogeneous_polynomial_evaluation_kernel(
    Tensor<scalar_t, 1> output, Tensor<scalar_t, 2> A, Tensor<scalar_t, 1> C, Tensor<int32_t, 2> indices_A
) {
    extern __shared__ char buffer[];

    int32_t nbatch = A.shape[0];
    int32_t nnu1 = A.shape[1];
    int32_t nbasis = C.shape[0];

    void* sptr = buffer;
    size_t space = 0;

    /* shared buffers */
    scalar_t* buffer_nu1 = shared_array<scalar_t>(nnu1, sptr, &space);
    scalar_t* tmp_sum = shared_array<scalar_t>(NWARPS_PER_BLOCK, sptr, &space);
    scalar_t* buffer_indices_A =
        shared_array<scalar_t>((blockDim.x + 1) * polynomial_order, sptr, &space);

    int32_t batch_id = blockIdx.x;

    if (batch_id > nbatch) {
        return;
    }

    // load all of A into shared memory
    for (int32_t i = threadIdx.x; i < nnu1; i += blockDim.x) {
        buffer_nu1[i] = A.data[batch_id * nnu1 + i];
    }

    __syncthreads();

    scalar_t batch_sum = 0.0;
    scalar_t c = 0.0; // kahans summation

    if (threadIdx.x == 0) {
        output.data[batch_id] = 0.0;
    }

    // indices_A : nbasis, polynomial_order
    for (int32_t i = 0; i < nbasis; i += blockDim.x) {

        __syncthreads();

        int32_t i_monomial =
            threadIdx.x % polynomial_order;         // [0 -> polynomial_order] : indices_A[*, :]
        int32_t x = threadIdx.x / polynomial_order; // [0 -> nx] -> indices_A[:, *]
        int32_t nx = find_integer_divisor(blockDim.x, polynomial_order);

        for (int32_t ii = x; ii < blockDim.x; ii += nx) {
            buffer_indices_A[i_monomial * blockDim.x + ii] =
                indices_A.data[i * polynomial_order + ii * polynomial_order + i_monomial];
        }

        __syncthreads();

        int32_t basis = i + threadIdx.x;

        if (basis < nbasis) {

            // need to load blockDim.x * polynomial_order elements into shared memory first

            scalar_t tmp = 1.0;

#pragma unroll
            for (int32_t i_monomial = 0; i_monomial < polynomial_order; i_monomial++) {
                int32_t idx = buffer_indices_A
                    [i_monomial * blockDim.x + threadIdx.x]; // indices_A.data[i_monomial
                                                             // * indices_A.shape[0] + basis];

                tmp *= buffer_nu1[idx];
            }

            scalar_t y = tmp * C.data[basis] - c;
            scalar_t t = batch_sum + y;
            c = (t - batch_sum) - y;
            batch_sum = t;
        }
    }

    for (int32_t offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        batch_sum += __shfl_down_sync(FULL_MASK, batch_sum, offset);
    }

    if (threadIdx.x % WARP_SIZE == 0) {
        tmp_sum[threadIdx.x / WARP_SIZE] = batch_sum;
    }

    __syncthreads();

    scalar_t out = 0.0;

    if (threadIdx.x == 0) {
        for (int32_t i = 0; i < blockDim.x / WARP_SIZE; i++) {
            out += tmp_sum[i];
        }

        output.data[batch_id] = out;
    }
}

template <typename scalar_t>
void mops::cuda::homogeneous_polynomial_evaluation(
    Tensor<scalar_t, 1> output, Tensor<scalar_t, 2> A, Tensor<scalar_t, 1> C, Tensor<int32_t, 2> indices_A
) {

    int32_t nbatch = output.shape[0];
    int32_t nnu1 = A.shape[1];
    size_t polynomial_order = indices_A.shape[1];

    dim3 block_dim(nbatch);

    dim3 thread_block(WARP_SIZE * NWARPS_PER_BLOCK, 1, 1);

    void* sptr = nullptr;
    size_t space = 0;

    shared_array<scalar_t>(nnu1, sptr, &space);
    shared_array<scalar_t>(thread_block.x / WARP_SIZE, sptr, &space);
    shared_array<int32_t>((thread_block.x + 1) * polynomial_order, sptr, &space);

    if (polynomial_order <= 10) {
        switch (polynomial_order) {
        case 0:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 0>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 1:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 1>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 2:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 2>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 3:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 3>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 4:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 4>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 5:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 5>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 6:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 6>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 7:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 7>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 8:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 8>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 9:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 9>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        case 10:
            homogeneous_polynomial_evaluation_kernel<scalar_t, 10>
                <<<block_dim, thread_block, space>>>(output, A, C, indices_A);
            break;
        default:
            break;
        }
    }

    CUDA_CHECK_ERROR(hipGetLastError());

    CUDA_CHECK_ERROR(hipDeviceSynchronize());
}

// explicit instanciations of CUDA templates
template void mops::cuda::homogeneous_polynomial_evaluation<float>(
    Tensor<float, 1> output, Tensor<float, 2> A, Tensor<float, 1> C, Tensor<int32_t, 2> indices_A
);

template void mops::cuda::homogeneous_polynomial_evaluation<double>(
    Tensor<double, 1> output, Tensor<double, 2> A, Tensor<double, 1> C, Tensor<int32_t, 2> indices_A
);

template <typename scalar_t, int32_t polynomial_order>
__global__ void homogeneous_polynomial_evaluation_vjp_kernel(
    Tensor<scalar_t, 2> grad_A,
    Tensor<scalar_t, 1> grad_output,
    Tensor<scalar_t, 2> A,
    Tensor<scalar_t, 1> C,
    Tensor<int32_t, 2> indices_A

) {
    extern __shared__ char buffer[];

    int32_t nbatch = A.shape[0];
    int32_t nnu1 = A.shape[1];
    int32_t nbasis = C.shape[0];

    void* sptr = buffer;
    size_t space = 0;

    /* shared buffers */
    scalar_t* buffer_nu1 = shared_array<scalar_t>(nnu1, sptr, &space);
    scalar_t* buffer_gradA = shared_array<scalar_t>(nnu1, sptr, &space);
    scalar_t* buffer_indices_A =
        shared_array<scalar_t>((blockDim.x + 1) * polynomial_order, sptr, &space);

    int32_t batch_id = blockIdx.x;

    if (batch_id > nbatch) {
        return;
    }

    // load all of A into shared memory
    for (int32_t i = threadIdx.x; i < nnu1; i += blockDim.x) {
        buffer_nu1[i] = A.data[batch_id * nnu1 + i];
        buffer_gradA[i] = 0.0;
    }

    __syncthreads();

    scalar_t gout = grad_output.data[batch_id];

    // indices_A : nbasis, polynomial_order
    for (int32_t i = 0; i < nbasis; i += blockDim.x) {

        __syncthreads();

        int32_t i_monomial =
            threadIdx.x % polynomial_order;         // [0 -> polynomial_order] : indices_A[*, :]
        int32_t x = threadIdx.x / polynomial_order; // [0 -> nx] -> indices_A[:, *]
        int32_t nx = find_integer_divisor(blockDim.x, polynomial_order);

        for (int32_t ii = x; ii < blockDim.x; ii += nx) {
            buffer_indices_A[i_monomial * blockDim.x + ii] =
                indices_A.data[i * polynomial_order + ii * polynomial_order + i_monomial];
        }

        __syncthreads();

        int32_t basis = i + threadIdx.x;

        if (basis < nbasis) {

            scalar_t c = C.data[basis] * gout;

            for (int32_t i_monomial = 0; i_monomial < polynomial_order; i_monomial++) {

                scalar_t tmp_i = c;

                for (int32_t j_monomial = 0; j_monomial < polynomial_order; j_monomial++) {

                    if (i_monomial == j_monomial) {
                        continue;
                    }

                    int32_t idx_j =
                        buffer_indices_A[j_monomial * blockDim.x + threadIdx.x]; // indices_A.data[j_monomial
                                                                                 // * indices_A.shape[0] + basis];

                    tmp_i *= buffer_nu1[idx_j];
                }

                int32_t idx_i = buffer_indices_A[i_monomial * blockDim.x + threadIdx.x];

                atomicAdd(&buffer_gradA[idx_i], tmp_i);
            }
        }
    }

    __syncthreads();

    for (int32_t i = threadIdx.x; i < nnu1; i += blockDim.x) {
        grad_A.data[batch_id * nnu1 + i] = buffer_gradA[i];
    }
}

template <typename scalar_t>
void mops::cuda::homogeneous_polynomial_evaluation_vjp(
    Tensor<scalar_t, 2> grad_A,
    Tensor<scalar_t, 1> grad_output,
    Tensor<scalar_t, 2> A,
    Tensor<scalar_t, 1> C,
    Tensor<int32_t, 2> indices_A
) {

    int32_t nbatch = grad_output.shape[0];
    int32_t nnu1 = A.shape[1];
    size_t polynomial_order = indices_A.shape[1];

    dim3 block_dim(nbatch);

    dim3 thread_block(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);

    void* sptr = nullptr;
    size_t space = 0;

    shared_array<scalar_t>(2 * nnu1, sptr, &space);
    shared_array<int32_t>((thread_block.x + 1) * polynomial_order, sptr, &space);

    if (polynomial_order <= 10) {
        switch (polynomial_order) {
        case 0:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 0>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 1:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 1>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 2:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 2>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 3:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 3>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 4:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 4>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 5:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 5>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 6:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 6>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 7:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 7>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 8:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 8>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 9:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 9>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        case 10:
            homogeneous_polynomial_evaluation_vjp_kernel<scalar_t, 10>
                <<<block_dim, thread_block, space>>>(grad_A, grad_output, A, C, indices_A);
            break;
        default:
            break;
        }
    }

    CUDA_CHECK_ERROR(hipGetLastError());

    CUDA_CHECK_ERROR(hipDeviceSynchronize());
}

// explicit instanciations of CUDA templates
template void mops::cuda::homogeneous_polynomial_evaluation_vjp<float>(
    Tensor<float, 2> gradA,
    Tensor<float, 1> grad_output,
    Tensor<float, 2> A,
    Tensor<float, 1> C,
    Tensor<int32_t, 2> indices_A
);

template void mops::cuda::homogeneous_polynomial_evaluation_vjp<double>(
    Tensor<double, 2> gradA,
    Tensor<double, 1> grad_output,
    Tensor<double, 2> A,
    Tensor<double, 1> C,
    Tensor<int32_t, 2> indices_A
);