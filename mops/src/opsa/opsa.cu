#include "hip/hip_runtime.h"

#include "mops/cuda_utils.cuh"
#include "mops/opsa_cuda.cuh"

using namespace mops::cuda;

#define WARP_SIZE 32
#define NWARPS_PER_BLOCK 4

template <typename scalar_t, const int32_t TA, const int32_t TB>
__device__ void outer_product_scatter_add_kernel(
    const scalar_t *__restrict__ A, // [nedges, nfeatures_A]
    const scalar_t *__restrict__ B, // [nedges, nfeatures_B]
    const int32_t nnodes,           // number of nodes we're summing into
    const int32_t nedges_total,     // number of edges -> batch size of A and B
    const int32_t nfeatures_A,      // number of features of A
    const int32_t nfeatures_B,      // number of features of B
    const int32_t
        *__restrict__ first_occurences, // indices in indices_output where the
                                        // values change [nnodes]
    const int32_t *__restrict__ indices_output, // sorted list of indices to sum
                                                // into [nedges]
    scalar_t
        *__restrict__ output // shape: [nnodes, nfeatures_B, nfeatures_A] ->
                             // this ordering because contiguity of threadCol
) {

    extern __shared__ char buffer[];

    const int32_t threadCol = threadIdx.x % WARP_SIZE;
    const int32_t threadRow = threadIdx.x / WARP_SIZE;
    const int32_t nThreadRow = blockDim.x / WARP_SIZE;

    /* registers to hold components of A, B and output - used to increase
     * arithmetic intensity.
     */
    scalar_t regA[TA] = {0.0};
    scalar_t regB[TB] = {0.0};
    scalar_t regOP[TA * TB] = {0.0};

    const int32_t edge_start = first_occurences[blockIdx.x];
    const int32_t edge_end = (blockIdx.x == nnodes - 1)
                                 ? nedges_total
                                 : first_occurences[blockIdx.x + 1];
    const int32_t node_index = indices_output[edge_start];
    const int32_t nedges = edge_end - edge_start;

    /* total number of columns of A we can process is TA * WARP_SIZE, so
     * we need to loop find_integer_divisor(nfeatures_A, TA*WARP_SIZE) times
     */

    int32_t niter_A = find_integer_divisor(nfeatures_A, TA * WARP_SIZE);
    int32_t niter_B = find_integer_divisor(nfeatures_B, TB * nThreadRow);

    for (int32_t iter_B = 0; iter_B < niter_B; iter_B++) {
        int32_t global_B = iter_B * TB * nThreadRow;

        for (int32_t iter_A = 0; iter_A < niter_A; iter_A++) {
            int32_t global_A = iter_A * TA * WARP_SIZE;

            /*
             *  clear registers
             */
            for (int32_t i = 0; i < TA; i++) {
                regA[i] = 0;
            }

            for (int32_t i = 0; i < TB; i++) {
                regB[i] = 0;
            }

            for (int32_t i = 0; i < TA * TB; i++) {
                regOP[i] = 0.0;
            }

            for (int32_t edge_idx = 0; edge_idx < nedges; edge_idx++) {

                int32_t edge = edge_idx + edge_start;

                /*
                 *  load A from GMEM into local registers
                 */
                for (int32_t i = 0; i < TA; i++) {

                    if (global_A + i * WARP_SIZE + threadCol < nfeatures_A)
                        regA[i] = A[edge * nfeatures_A + global_A +
                                    i * WARP_SIZE + threadCol];
                }

                /*
                 *  load B from GMEM into local registers
                 */
                for (int32_t i = 0; i < TB; i++) {
                    if (global_B + i * nThreadRow + threadRow < nfeatures_B)
                        regB[i] = B[edge * nfeatures_B + global_B +
                                    i * nThreadRow + threadRow];
                }

                /*
                 * perform outer product in registers
                 */
                for (int32_t j = 0; j < TB; j++) {
                    for (int32_t i = 0; i < TA; i++) {
                        regOP[j * TA + i] += regA[i] * regB[j];
                    }
                }
            }

            /*
             * writeout the content of regOP to the output for this block of
             * [node, nfeatures_A, nfeatures_B]
             */
            for (int32_t j = 0; j < TB; j++) {
                if (global_B + j * nThreadRow + threadRow < nfeatures_B) {
                    for (int32_t i = 0; i < TA; i++) {
                        if (global_A + i * WARP_SIZE + threadCol <
                            nfeatures_A) {
                            output[node_index * nfeatures_B * nfeatures_A +
                                   (global_B + j * nThreadRow + threadRow) *
                                       nfeatures_A +
                                   global_A + i * WARP_SIZE + threadCol] =
                                regOP[j * TA + i];
                        }
                    }
                }
            }
        }
    }
}
namespace mops::cuda {
template <typename scalar_t>
void outer_product_scatter_add_cuda(
    const scalar_t *__restrict__ A, // [nedges, nfeatures_A]
    const scalar_t *__restrict__ B, // [nedges, nfeatures_B]
    const int32_t nnodes,           // number of nodes we're summing into
    const int32_t nedges,           // number of edges -> batch size of A and B
    const int32_t nfeatures_A,      // number of features of A
    const int32_t nfeatures_B,      // number of features of B
    const int32_t
        *__restrict__ first_occurences, // indices in indices_output where the
                                        // values change [nnodes]
    const int32_t *__restrict__ indices_output, // sorted list of indices to sum
                                                // into [nedges]
    scalar_t
        *__restrict__ output // shape: [nnodes, nfeatures_B, nfeatures_A]
                             // -> this ordering because contiguity of threadCol

) {

    dim3 gridDim(nnodes, 1, 1);

    dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);

    outer_product_scatter_add_kernel<scalar_t, 4, 4><<<gridDim, blockDim, 0>>>(
        A, B, nnodes, nedges, nfeatures_A, nfeatures_B, first_occurences,
        indices_output, output);

    hipDeviceSynchronize();
}

} // namespace mops::cuda
