#include "mops.hpp"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

/*host macro that checks for errors in CUDA calls, and prints the file + line
 * and error string if one occurs
 */
#define CUDA_CHECK(call)                                                                           \
    do {                                                                                           \
        hipError_t cudaStatus = (call);                                                           \
        if (cudaStatus != hipSuccess) {                                                           \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - "                  \
                      << hipGetErrorString(cudaStatus) << std::endl;                              \
            hipDeviceReset();                                                                     \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    } while (0)

int main() {
    // To avoid calls with a very large number of arguments,
    // mops uses a mops::Tensor<T, N_DIMS> struct which simply
    // consists a data pointer and a shape in the form of a std::array.
    //
    // All mops operations take mops::Tensor objects as their
    // inputs, and these can be initialized in the following way:

    auto A = std::vector<double>(100 * 20);
    auto B = std::vector<double>(100 * 5);
    auto indices_output = std::vector<int32_t>(100);
    auto output = std::vector<double>(10 * 20 * 5);

    double* A_cuda;
    double* B_cuda;
    int32_t* indices_output_cuda;
    double* output_cuda;

    CUDA_CHECK(hipMalloc(&A_cuda, A.size() * sizeof(double)));
    CUDA_CHECK(hipMalloc(&B_cuda, B.size() * sizeof(double)));
    CUDA_CHECK(hipMalloc(&indices_output_cuda, indices_output.size() * sizeof(int32_t)));
    CUDA_CHECK(hipMalloc(&output_cuda, output.size() * sizeof(double)));

    CUDA_CHECK(hipMemcpy(A_cuda, A.data(), A.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(B_cuda, B.data(), B.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        indices_output_cuda,
        indices_output.data(),
        indices_output.size() * sizeof(int32_t),
        hipMemcpyHostToDevice
    ));
    CUDA_CHECK(hipMemcpy(
        output_cuda, output.data(), output.size() * sizeof(double), hipMemcpyHostToDevice
    ));

    mops::cuda::outer_product_scatter_add<double>(
        {output_cuda, {100, 20, 5}},
        {A_cuda, {100, 20}},
        {B_cuda, {100, 5}},
        {indices_output_cuda, {100}}
    );

    CUDA_CHECK(hipMemcpy(
        output.data(), output_cuda, output.size() * sizeof(double), hipMemcpyDeviceToHost
    ));

    CUDA_CHECK(hipFree(A_cuda));
    CUDA_CHECK(hipFree(B_cuda));
    CUDA_CHECK(hipFree(indices_output_cuda));
    CUDA_CHECK(hipFree(output_cuda));

    return 0;
}
